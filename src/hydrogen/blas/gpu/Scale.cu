#include "hip/hip_runtime.h"
#include <hydrogen/blas/gpu/Scale.hpp>

#include <El/hydrogen_config.h>
#include <hydrogen/meta/TypeTraits.hpp>
#ifdef HYDROGEN_HAVE_CUDA
#include <hydrogen/device/gpu/CUDA.hpp>
#include <hip/hip_runtime.h>
#elif defined(HYDROGEN_HAVE_ROCM)
#include <hydrogen/device/gpu/ROCm.hpp>
#include <hip/hip_runtime.h>
#endif

namespace
{

template <typename T, typename SizeT>
__global__ void scale_1d_kernel_naive(
    SizeT num_entries, T alpha, T* A, SizeT stride_A)
{
    SizeT const idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < num_entries)
        A[idx*stride_A] *= alpha;
}

template <int TILE_DIM, int BLK_COLS, typename T, typename SizeT>
__global__ void scale_2d_kernel_naive(
    SizeT m, SizeT n, T alpha, T* A, SizeT lda)
{
    size_t const row_idx = blockIdx.x * TILE_DIM + threadIdx.x;
    size_t const col_idx = blockIdx.y * TILE_DIM + threadIdx.y;

    if (row_idx < m)
    {
        for (int ii = 0; ii < TILE_DIM && col_idx + ii < n; ii += BLK_COLS)
            A[row_idx + (col_idx+ii)*lda] *= alpha;
    }
}

}// namespace <anon>

namespace hydrogen
{

template <typename T, typename SizeT, typename>
void Scale_GPU_impl(
    SizeT num_entries,
    T const& alpha_in, T* A, SizeT lda,
    SyncInfo<Device::GPU> const& sync_info)
{
    if (!num_entries)
        return;

    constexpr size_t threads_per_block = 128;
    auto blocks = (num_entries + threads_per_block - 1)/ threads_per_block;
    NativeGPUType<T> alpha = *AsNativeGPUType(&alpha_in);
    gpu::LaunchKernel(
        scale_1d_kernel_naive<NativeGPUType<T>, SizeT>,
        blocks, threads_per_block, 0, sync_info,
        num_entries, alpha, AsNativeGPUType(A), lda);
}

template <typename T, typename SizeT, typename>
void Scale_GPU_impl(
    SizeT num_rows, SizeT num_cols,
    T const& alpha_in, T* A, SizeT lda,
    SyncInfo<Device::GPU> const& sync_info)
{
    if (num_rows == TypeTraits<SizeT>::Zero()
        || num_cols == TypeTraits<SizeT>::Zero())
    {
        return;
    }

    constexpr int TILE_DIM = 32;
    constexpr int BLK_COLS = 8;

    dim3 blks((num_rows + TILE_DIM - 1) / TILE_DIM,
              (num_cols + TILE_DIM - 1) / TILE_DIM,
              1);
    dim3 thds(TILE_DIM, BLK_COLS, 1);

    NativeGPUType<T> alpha = *AsNativeGPUType(&alpha_in);
    gpu::LaunchKernel(
        scale_2d_kernel_naive<TILE_DIM,BLK_COLS,NativeGPUType<T>,SizeT>,
        blks, thds, 0, sync_info,
        num_rows, num_cols, alpha, AsNativeGPUType(A), lda);
}

#define ETI(DataType, SizeType)                         \
    template void Scale_GPU_impl(                       \
        SizeType,                                       \
        DataType const&, DataType*, SizeType,           \
        SyncInfo<Device::GPU> const&);                  \
    template void Scale_GPU_impl(                       \
        SizeType, SizeType,                             \
        DataType const&, DataType*, SizeType,           \
        SyncInfo<Device::GPU> const&)

#define ETI_ALL_SIZE_TYPES(ScalarT)               \
    ETI(ScalarT, int);                            \
    ETI(ScalarT, long);                           \
    ETI(ScalarT, long long);                      \
    ETI(ScalarT, unsigned);                       \
    ETI(ScalarT, size_t)

#ifdef HYDROGEN_GPU_USE_FP16
ETI_ALL_SIZE_TYPES(gpu_half_type);
#endif

ETI_ALL_SIZE_TYPES(float);
ETI_ALL_SIZE_TYPES(double);
ETI_ALL_SIZE_TYPES(El::Complex<float>);
ETI_ALL_SIZE_TYPES(El::Complex<double>);

}// namespace hydrogen
